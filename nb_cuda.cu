#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/extrema.h>
#include <assert.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <cfloat>
#include "io.hpp"
#include "evaluate.h"

#define CUDA_CHECK_RETURN(value) { \
               hipError_t _m_cudaStat = value;\
               if (_m_cudaStat != hipSuccess) {\
                       fprintf(stderr, "Error %s at line %d in file %s\n",\
                                       hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);\
                                       exit(1);\
               }}
#define SIZE_TRAIN 128

double get_wall_time(){
    struct timeval time;
    if (gettimeofday(&time,NULL)){
        //  Handle error
        return 0;
    }
    return (double)time.tv_sec + (double)time.tv_usec * .000001;
}


__global__ void trainning_kernel2(int *freqClassVector, double *matrixTermFreq,
		double* totalFreqClassVector, int *docTestIndexVector,
		int *docTestVector, double *docTestFreqVector, int *docClasse,
		int numClasses, int numTerms, int numDocsTest, double *freqTermVector,
		double totalTermFreq, int totalTerms, double lambda, double alpha,
		int numDocs) {

	int vecs, len, term, bestClass;
	double freq;
	double prob, nt, highestProb;
	extern __shared__ double temp[]; // used to hold segment of the vector (size nthreads)
	// plus 3 integers (vecs, len, partial sum) at the end
	int tid = threadIdx.x;

	if (tid == 0) {  // thread 0 calculates vecs and len
		//vecs - vector size
		temp[blockDim.x + 1] = (docTestIndexVector[blockIdx.x + 1]
				- docTestIndexVector[blockIdx.x]);
		// len - number of segments (size nthreads) of the vector
		if(temp[blockDim.x + 1] > blockDim.x)
			temp[blockDim.x + 2] = ceil(temp[blockDim.x + 1] / (float) blockDim.x);
		else
			temp[blockDim.x + 2] = 1.0;
		// partial sum initialization
		//temp[blockDim.x + 3] = 0.0;
	}
	__syncthreads();

	vecs = temp[blockDim.x + 1]; // communicate vecs and len's values to other threads
	len = (int) temp[blockDim.x + 2];

	for (int c = 0; c < numClasses; c++) {
		if (tid == 0) {
			// partial sum initialization
			temp[blockDim.x + 3] = log(
					(freqClassVector[c] + alpha)
							/ (numDocs + alpha * numClasses));
		}
		__syncthreads();
		for (int b = 0; b < len; b++) { // loop through 'len' segments
			// first, each thread loads data into shared memory
			if ((b * blockDim.x + tid) >= vecs) // check if outside 'vec' boundary
				temp[tid] = 0.0;
			else {
				term = docTestVector[docTestIndexVector[blockIdx.x]
						+ b * blockDim.x + tid];
				if(freqTermVector[term] != 0){
					freq = docTestFreqVector[docTestIndexVector[blockIdx.x]
						+ b * blockDim.x + tid];
					prob = (matrixTermFreq[c * numTerms + term] + alpha)
						/ (totalFreqClassVector[c] + alpha * totalTerms);
					nt = freqTermVector[term] / totalTermFreq;
					prob = lambda * nt + (1.0 - lambda) * prob;
					temp[tid] = freq * log(prob);
				}
				else{
					temp[tid] = 0.0;
				}
			}
			__syncthreads();

			// next, perform binary tree reduction on shared memory
			for (int d = blockDim.x >> 1; d > 0; d >>= 1) {
				if (tid < d)
					temp[tid] += (tid + d) >= vecs ? 0.0 : temp[tid + d];
				__syncthreads();
			}

			// first thread puts partial result into shared memory
			if (tid == 0) {
				temp[blockDim.x + 3] += temp[0];
			}
			__syncthreads();
		}
		// finally, first thread puts result into global memory
		if (tid == 0) {
			if (c == 0) {
				highestProb = temp[blockDim.x + 3];
				bestClass = 0;
			} else if (temp[blockDim.x + 3] > highestProb) {
				highestProb = temp[blockDim.x + 3];
				bestClass = c;
			}
		}
		__syncthreads();
	}
	if (tid == 0) {
		docClasse[blockIdx.x] = bestClass;
	}
}

extern "C"{
void nb_gpu(const char* filenameTreino, const char* filenameTeste,
		int numDocs, int numClasses, int numTerms, int numDocsTest,
		int numTermsTest, double alpha, double lambda, int cudaDevice) {

	double begin, end;
	begin=get_wall_time();
	int block_size, n_blocks;
	int *docTestIndexVector = (int*) malloc((numDocsTest + 1) * sizeof(int));
	int *docTestVector = NULL;
	double *docTestFreqVector = NULL;

	int *freqClassVector = (int*) malloc(numClasses * sizeof(int));
	double *totalFreqClassVector = (double*) malloc(
			numClasses * sizeof(double));
	double *matrixTermFreq = (double*) malloc(
			(numTerms * numClasses) * sizeof(double));
	double *freqTermVector = (double*) malloc((numTerms) * sizeof(double));
	double totalTermFreq = 0.0;
	int totalTerms = 0;

	for (int i = 0; i < numClasses; i++) {
		totalFreqClassVector[i] = 0.0;
		freqClassVector[i] = 0;
		for (int j = 0; j < numTerms; j++) {
			matrixTermFreq[i * numTerms + j] = 0.0;
		}
	}
	for (int j = 0; j < numTerms; j++) {
		freqTermVector[j] = 0.0;
	}
	
	readTrainData(filenameTreino, freqClassVector, totalFreqClassVector, freqTermVector, &totalTermFreq, 
		numClasses, numTerms, &totalTerms, matrixTermFreq);

	double *matrixTermFreq_D;
	hipMalloc((void **) &matrixTermFreq_D,
			sizeof(double) * (numTerms * numClasses));
	hipMemcpy(matrixTermFreq_D, matrixTermFreq,
			sizeof(double) * (numTerms * numClasses), hipMemcpyHostToDevice);

	int *freqClassVector_D;
	hipMalloc((void **) &freqClassVector_D, sizeof(int) * numClasses);
	hipMemcpy(freqClassVector_D, freqClassVector, sizeof(int) * numClasses,
			hipMemcpyHostToDevice);

	double *totalFreqClassVector_D;
	hipMalloc((void **) &totalFreqClassVector_D, sizeof(double) * numClasses);
	hipMemcpy(totalFreqClassVector_D, totalFreqClassVector,
			sizeof(double) * numClasses, hipMemcpyHostToDevice);

	double *freqTermVector_D;
	hipMalloc((void **) &freqTermVector_D, sizeof(double) * numTerms);
	hipMemcpy(freqTermVector_D, freqTermVector, sizeof(double) * numTerms,
			hipMemcpyHostToDevice);

	int *realClass = (int*) malloc((numDocsTest + 1) * sizeof(int));

	
	docTestVector = readTestData(filenameTeste, docTestIndexVector, realClass,
			&docTestFreqVector);

	end=get_wall_time();
	cerr << "read test time " << end - begin << endl;
	begin=get_wall_time();

	int *docTestIndexVector_D;
	hipMalloc((void **) &docTestIndexVector_D,
			sizeof(int) * (numDocsTest + 1));
	hipMemcpy(docTestIndexVector_D, docTestIndexVector,
			sizeof(int) * (numDocsTest + 1), hipMemcpyHostToDevice);

	int *docTestVector_D;
	hipMalloc((void **) &docTestVector_D,
			sizeof(int) * docTestIndexVector[numDocsTest]);
	hipMemcpy(docTestVector_D, docTestVector,
			sizeof(int) * docTestIndexVector[numDocsTest],
			hipMemcpyHostToDevice);
	
	double *docTestFreqVector_D;
	hipMalloc((void **) &docTestFreqVector_D,
			sizeof(double) * docTestIndexVector[numDocsTest]);
	hipMemcpy(docTestFreqVector_D, docTestFreqVector,
			sizeof(double) * docTestIndexVector[numDocsTest],
			hipMemcpyHostToDevice);

	int *docClasse = (int*) malloc((numDocsTest) * sizeof(int));	

	int *docClasse_D;
	hipError_t status = hipHostMalloc((void **) &docClasse_D, sizeof(int) * (numDocsTest));
    if (status != hipSuccess)
		printf("Error allocating pinned host memoryn");
    double *valor = (double*) malloc(2*sizeof(double));
   
    block_size = SIZE_TRAIN;
    n_blocks = numDocsTest;
    trainning_kernel2<<<n_blocks, block_size, (block_size + 3) * sizeof(double)>>>(
	  freqClassVector_D, matrixTermFreq_D, totalFreqClassVector_D,
	  docTestIndexVector_D, docTestVector_D, docTestFreqVector_D,
	  docClasse_D, numClasses, numTerms, numDocsTest, freqTermVector_D,
	  totalTermFreq, totalTerms, lambda, alpha, numDocs);

    hipMemcpy(docClasse, docClasse_D, sizeof(int) * (numDocsTest),
	  hipMemcpyDeviceToHost);

    valor[0] = evaluate(realClass, docClasse, numDocsTest, 1);
    valor[1] = evaluate(realClass, docClasse, numDocsTest, 0);

    std::cout << alpha << " " << lambda << " " << valor[0]*100 << " " << valor[1]*100 << std::endl;

	end=get_wall_time();
	cerr << "classification and evaluation times " << end - begin << endl;

	hipHostFree(docClasse_D);
	hipFree(docTestIndexVector_D);
	hipFree(docTestVector_D);
	hipFree(docTestFreqVector_D);
	hipFree(freqTermVector_D);
	hipFree(matrixTermFreq_D);
	hipFree(freqClassVector_D);
	hipFree(totalFreqClassVector_D);
	free(totalFreqClassVector);	
	free(matrixTermFreq);
	free(freqTermVector);
	free(freqClassVector);
	free(realClass);
	free(docTestIndexVector);
	free(docTestVector);
	free(docClasse);
	free(docTestFreqVector);
	free(valor);
	return;
}
}
